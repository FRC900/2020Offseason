#include "ros/console.h"
#include "base_trajectory/cuda_utils.h"

void cudaSafeCallWrapper(hipError_t err, const char* file, const int line)
{
	if (hipSuccess != err)
	{
		ROS_ERROR_STREAM("CUDA error : " <<
						 "\nFile: " << file <<
						 "\nLine Number: " << line <<
						 "\nReason:" << hipGetErrorString(err));
		//hipDeviceReset();
		//exit(EXIT_FAILURE);
	}
}

size_t numBlocks(size_t threads_per_block, size_t N)
{
	return (N + threads_per_block - 1) / threads_per_block;
}


bool hasCudaGPU(void)
{
	int deviceCount;
	hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
	if (cudaResultCode != hipSuccess)
		deviceCount = 0;
	/* machines with no GPUs can still report one emulation device */
	int gpuDeviceCount = 0;
	for (int device = 0; device < deviceCount; ++device) {
		struct hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, device);
		if (properties.major != 9999) /* 9999 means emulation only */
			++gpuDeviceCount;
	}

	return (gpuDeviceCount > 0);
}
